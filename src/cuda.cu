#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <chrono>
#include <fstream>
#include <iostream>
#include <sstream>
#include <vector>

#include "utils/hip/hip_runtime_api.h"
#include "utils/node.h"
#include "utils/utilities.h"

using std::chrono::duration;
using std::chrono::duration_cast;
using std::chrono::high_resolution_clock;
using std::chrono::milliseconds;

std::vector<Body *> bodies;
ParallelBody *cudaBodies;
ParallelBody *cudaBodies_d;

double theta = 0.8;
size_t maxStackSize = 0;
int numNodes = 0;

int NUM_THREADX = 64;
// Building a separate tree for access on cuda
// preprocessing

__host__ std::pair<Node *, ParallelNode *> CopyTreeToDevice() {
    Node *octree = new Node(bodies);
    octree->ConstructTree();
    // ParallelNode *cudaOctree_h;
    ParallelNode *cudaOctree_d;
    checkCudaErrors(hipMalloc(&cudaOctree_d, sizeof(ParallelNode)));

    std::pair<Node *, ParallelNode *> octree_pair = std::make_pair(octree, cudaOctree_d);

    std::stack<std::pair<Node *, ParallelNode *>> traversalStack;
    traversalStack.push(octree_pair);
    // traverse the original tree
    numNodes++;
    while (!traversalStack.empty()) {
        maxStackSize = std::max(traversalStack.size(), maxStackSize);
        std::pair<Node *, ParallelNode *> currNodePair = traversalStack.top();
        traversalStack.pop();

        if (currNodePair.first->GetNumBodies() < 1) continue;

        ParallelNode *cudaNode_h = new ParallelNode();
        ParallelNode *cudaNode_d = currNodePair.second;

        // copy the bare minimum info needed for barneshut to a Cuda Node

        cudaNode_h->centreOfMass[0] = currNodePair.first->GetCOM(true).x;
        cudaNode_h->centreOfMass[1] = currNodePair.first->GetCOM(true).y;
        cudaNode_h->centreOfMass[2] = currNodePair.first->GetCOM(true).z;
        cudaNode_h->numBodies = currNodePair.first->GetNumBodies();
        cudaNode_h->mass = currNodePair.first->GetMass();
        cudaNode_h->diagDist = std::sqrt(DistSquared(currNodePair.first->GetMaxBound(), currNodePair.first->GetMinBound()));

        if (currNodePair.first->GetNumBodies() != 1) {
            for (int i = 0; i < 8; i++) {
                // allocate each child on the device
                ParallelNode *child_d;
                checkCudaErrors(hipMalloc(&child_d, sizeof(ParallelNode)));

                // keep track of the address of children from the device on host
                cudaNode_h->deviceChildren[i] = child_d;
                currNodePair.first->devicechildren[i] = child_d;
                traversalStack.push(std::make_pair(currNodePair.first->GetChildren()[i], child_d));
                numNodes++;
            }
        };

        // copy the data over
        if (currNodePair.second == cudaOctree_d)
            checkCudaErrors(hipMemcpy(cudaOctree_d, cudaNode_h, sizeof(ParallelNode), hipMemcpyHostToDevice));
        else
            checkCudaErrors(hipMemcpy(cudaNode_d, cudaNode_h, sizeof(ParallelNode), hipMemcpyHostToDevice));

        delete cudaNode_h;
    }

    return octree_pair;
}

void FreeTree(Node *octree, ParallelNode *cudaOctree_d) {
    std::stack<Node *> traversalStack;

    traversalStack.push(octree);

    while (!traversalStack.empty()) {
        Node *currNode = traversalStack.top();
        traversalStack.pop();
        if (currNode->GetNumBodies() <= 1) continue;

        for (int i = 0; i < 8; i++) {
            // clean up gpu memory
            ParallelNode *deviceChild = currNode->devicechildren[i];
            checkCudaErrors(hipFree(deviceChild));

            traversalStack.push(currNode->GetChildren()[i]);
        }
    }

    checkCudaErrors(hipFree(cudaOctree_d));
    delete octree;
}

void UpdateBodies(ParallelBody *cudaBodies) {
    for (int i = 0; i < bodies.size(); i++) {
        // std::cout<< cudaBodies[i].position[0]<<" "<<std::endl;
        bodies[i]->position.x = cudaBodies[i].position[0];
        bodies[i]->position.y = cudaBodies[i].position[1];
        bodies[i]->position.z = cudaBodies[i].position[2];

        bodies[i]->velocity.x = cudaBodies[i].velocity[0];
        bodies[i]->velocity.y = cudaBodies[i].velocity[1];
        bodies[i]->velocity.z = cudaBodies[i].velocity[2];
    }
}

__device__ double DistSquared_d(double *p1, double *p2) {
    double dist = 0.0;
    for (int i = 0; i < 3; i++) {
        dist += (p1[i] - p2[i]) * (p1[i] - p2[i]);
    }
    return dist;
}

__device__ bool isEqual(double one, double two) {
    double eps = 1.0;
    return (fabs(one - two) < eps);
}

__device__ bool SameBody(ParallelBody *body, ParallelNode *node) {
    if (node->numBodies != 1) return false;
    if (!isEqual(body->mass, node->mass)) return false;
    for (int i = 0; i < 3; i++) {
        if (!isEqual(body->position[i], node->centreOfMass[i])) return false;
    }

    return true;
}

__device__ void CalcuateForce(ParallelBody *body, ParallelNode *node, double *totalForce) {
    // TODO: if no work, check G
    double dist = DistSquared_d(body->position, node->centreOfMass);

    double magnitude = G * body->mass * node->mass * (1 / dist);
    double force[3];

    dist = sqrt(dist);
    for (int i = 0; i < 3; i++) {
        force[i] = magnitude * (node->centreOfMass[i] - body->position[i]) / dist;
    }

    for (int i = 0; i < 3; i++)
        totalForce[i] += force[i];
}

__global__ void BarnesHutKernel(ParallelNode *octree_d, size_t maxStackSize_d, ParallelBody *bodies_d, int N, double theta) {
    int threadID = blockIdx.x * blockDim.x + threadIdx.x;

    if (threadID >= N) return;

    ParallelBody *body = (ParallelBody *)bodies_d + threadID;
    double totalForce[3] = {0, 0, 0};

    extern __shared__ ParallelNode *traversalStack[];
    int stackIndex = threadIdx.x * maxStackSize_d;
    int topIndex = -1;  // index of the top element in the stack

    // pushing to stack
    topIndex += 1;
    traversalStack[topIndex + stackIndex] = octree_d;

    int counter = 0;
    // output_d[1] = 420.0;
    while (topIndex != -1) {
        // peeking and popping
        ParallelNode *currNode = traversalStack[topIndex + stackIndex];
        topIndex--;

        if (SameBody(body, currNode)) continue;
        double d = sqrt(DistSquared_d(body->position, currNode->centreOfMass));
        double s = currNode->diagDist;

        if ((s / d) > theta && currNode->numBodies != 1) {
            for (int i = 0; i < 8; i++) {
                ParallelNode *child = currNode->deviceChildren[i];

                if (child->numBodies != 0) {
                    topIndex++;
                    traversalStack[topIndex + stackIndex] = child;
                }
            }
        } else {
            CalcuateForce(body, currNode, totalForce);
        }
    }

    for (int i = 0; i < 3; i++) {
        body->velocity[i] += totalForce[i] / body->mass;
        body->position[i] += body->velocity[i];
    }
}

void BarnesHutCuda(int N, int numIterations) {
    std::pair<Node *, ParallelNode *> octreePair;

    checkCudaErrors(hipMalloc(&cudaBodies_d, sizeof(ParallelBody) * N));

    for (int i = 0; i < numIterations; i++) {
        std::pair<Node *, ParallelNode *> octreePair = CopyTreeToDevice();
        checkCudaErrors(hipMemcpy(cudaBodies_d, cudaBodies, sizeof(ParallelBody) * N, hipMemcpyHostToDevice));

        dim3 dimBlock(NUM_THREADX, 1, 1);
        dim3 dimGrid((bodies.size() + NUM_THREADX) / NUM_THREADX, 1, 1);

        BarnesHutKernel<<<dimGrid, dimBlock, maxStackSize * NUM_THREADX * sizeof(ParallelNode *)>>>(octreePair.second, maxStackSize, cudaBodies_d, N, theta);
        checkCudaErrors(hipDeviceSynchronize());

        checkCudaErrors(hipMemcpy(cudaBodies, cudaBodies_d, sizeof(ParallelBody) * N, hipMemcpyDeviceToHost));
        // FreeTree(octreePair.first, octreePair.second);
        delete octreePair.first;
        UpdateBodies(cudaBodies);
    }
}

int main(int argc, char *argv[]) {
    /* 1 argument
     * FILENAME: The file name to read data from
     */
    if (argc != 3) {
        std::cout << "Incorrect number of parameters given:"
                  << " 2 required, " << argc - 1 << " given" << std::endl
                  << "FILENAME: The file name to read data from" << std::endl
                  << "I: Number of iterations to run the simulations for" << std::endl;
        return 0;
    }
    std::string filename(argv[1]);
    int numIterations = atoi(argv[2]);

    // open the file
    std::ifstream inFile("../data/" + filename);

    // the number of particles, magnitude of the position,magnitude of the mass
    int N, P, M;
    inFile >> N >> P >> M;
    inFile.ignore();

    // helper variables for reading input
    std::string line;
    double mass;
    double x, y, z;

    bodies.reserve(N);
    cudaBodies = new ParallelBody[N];
    for (int i = 0; i < N; i++) {
        // read each body
        std::getline(inFile, line);

        // create a string stream
        std::stringstream ss(line);

        ss >> mass >> x >> y >> z;

        vec3 position(x, y, z);

        bodies.push_back(new Body(mass, position));
        bodies[i]->velocity = bodies[i]->position / std::pow(10, P - 3);
        bodies[i]->velocity.rot90z();

        cudaBodies[i].position[0] = position.x;
        cudaBodies[i].position[1] = position.y;
        cudaBodies[i].position[2] = position.z;
        cudaBodies[i].velocity[0] = bodies[i]->velocity.x;
        cudaBodies[i].velocity[1] = bodies[i]->velocity.y;
        cudaBodies[i].velocity[2] = bodies[i]->velocity.z;
        cudaBodies[i].mass = mass;
    }
    inFile.close();

    auto t1 = high_resolution_clock::now();
    BarnesHutCuda(N, numIterations);
    auto t2 = high_resolution_clock::now();
    auto ms_int = duration_cast<milliseconds>(t2 - t1);
    std::cout << 1.0 * ms_int.count() / numIterations << "ms  average for " << numIterations << " iterations, cuda\n";

    std::ofstream outFile("../out/cuda-" + filename);
    outFile.precision(15);

    outFile << N << std::endl;
    outFile << std::scientific;
    for (Body *body : bodies) {
        outFile << body->mass
                << " " << body->position.x
                << " " << body->position.y
                << " " << body->position.z
                << std::endl;
        delete body;
    }

    outFile.close();

    checkCudaErrors(hipFree(cudaBodies_d));
    delete[] cudaBodies;
    checkCudaErrors(hipDeviceReset());
}