#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <chrono>
#include <fstream>
#include <iostream>
#include <sstream>
#include <vector>

#include "OpenGL/gl_utilities.h"
#include "utils/hip/hip_runtime_api.h"
#include "utils/node.h"
#include "utils/utilities.h"

std::vector<Body *> bodies;
double theta = 0.8;
size_t maxStackSize = 0;
int numNodes = 0;

std::stringstream data("6 6 15 8.401877171547095e+15 -2.032763069026615e+04 -1.811363407102262e+05 1.140483239660925e+05 3.943829268190930e+15 -1.791981915863382e+05 5.544912736748732e+03 1.241392662871304e+05 7.830992237586059e+15 5.601020441137095e+03 -8.777286769754969e+04 7.708867393003816e+04 7.984400334760733e+15 3.344990699950628e+04 3.541127793886492e+05 6.195353842703000e+04 9.116473579367844e+15 1.701604548859206e+05 1.305680358210422e+05 -2.130698223755301e+04 1.000000000000000e+19 0.000000000000000e+00 0.000000000000000e+00 0.000000000000000e+00");
int NUM_THREADX = 1;
// Building a separate tree for access on cuda
// preprocessing
struct CudaNode {
    double centreOfMass[3];
    double mass;
    int numBodies;
    CudaNode *deviceChildren[8];
};

template <typename T>
void DebugPrint(T info) {
    std::cout << info << std::endl;
}

__host__ std::pair<Node *, CudaNode *> CopyTreeToDevice() {
    Node *octree = new Node(bodies);
    octree->ConstructTree();
    // CudaNode *cudaOctree_h;
    CudaNode *cudaOctree_d;
    checkCudaErrors(hipMalloc(&cudaOctree_d, sizeof(CudaNode)));

    std::pair<Node *, CudaNode *> octree_pair = std::make_pair(octree, cudaOctree_d);

    std::stack<std::pair<Node *, CudaNode *>> traversalStack;
    traversalStack.push(octree_pair);
    // traverse the original tree
    numNodes++;
    while (!traversalStack.empty()) {
        maxStackSize = std::max(traversalStack.size(), maxStackSize);
        std::pair<Node *, CudaNode *> currNodePair = traversalStack.top();
        traversalStack.pop();

        if (currNodePair.first->GetNumBodies() < 1) continue;

        CudaNode *cudaNode_h = new CudaNode();
        CudaNode *cudaNode_d = currNodePair.second;

        // copy the bare minimum info needed for barneshut to a Cuda Node

        cudaNode_h->centreOfMass[0] = currNodePair.first->GetCOM(false).x;
        cudaNode_h->centreOfMass[1] = currNodePair.first->GetCOM(false).y;
        cudaNode_h->centreOfMass[2] = currNodePair.first->GetCOM(false).z;
        cudaNode_h->numBodies = currNodePair.first->GetNumBodies();
        cudaNode_h->mass = currNodePair.first->GetMass();

        if (currNodePair.first->GetNumBodies() != 1) {
            for (int i = 0; i < 8; i++) {
                // allocate each child on the device
                CudaNode *child_d;
                checkCudaErrors(hipMalloc(&child_d, sizeof(CudaNode)));

                // keep track of the address of children from the device on host
                cudaNode_h->deviceChildren[i] = child_d;
                currNodePair.first->devicechildren[i] = child_d;
                traversalStack.push(std::make_pair(currNodePair.first->GetChildren()[i], child_d));
                numNodes++;
            }
        };

        // copy the data over
        if (currNodePair.second == cudaOctree_d)
            checkCudaErrors(hipMemcpy(cudaOctree_d, cudaNode_h, sizeof(CudaNode), hipMemcpyHostToDevice));
        else
            checkCudaErrors(hipMemcpy(cudaNode_d, cudaNode_h, sizeof(CudaNode), hipMemcpyHostToDevice));

        delete cudaNode_h;
    }

    return octree_pair;
}

void FreeDeviceTree(Node *octree, CudaNode *cudaOctree_d) {
    std::stack<Node *> traversalStack;

    traversalStack.push(octree);

    while (!traversalStack.empty()) {
        Node *currNode = traversalStack.top();
        traversalStack.pop();
        if (currNode->GetNumBodies() <= 1) continue;

        for (int i = 0; i < 8; i++) {
            // clean up gpu memory
            CudaNode *deviceChild = currNode->devicechildren[i];
            checkCudaErrors(hipFree(deviceChild));

            traversalStack.push(currNode->GetChildren()[i]);
        }
    }

    checkCudaErrors(hipFree(cudaOctree_d));
}

__global__ void BarnesHutKernel(CudaNode *octree_d, size_t maxStackSize_d, double *output_d) {
    extern __shared__ CudaNode *traversalStack[];
    int stackIndex = threadIdx.x * maxStackSize_d;
    int topIndex = -1;  // index of the top element in the stack

    // pushing to stack
    topIndex += 1;
    traversalStack[topIndex + stackIndex] = octree_d;

    int counter = 0;
    // output_d[1] = 420.0;
    while (topIndex != -1) {
        // peeking and popping
        CudaNode *currNode = traversalStack[topIndex + stackIndex];
        topIndex--;

        if (currNode->numBodies == 0) continue;
        output_d[counter] = currNode->mass;
        counter++;
        if (currNode->numBodies > 1) {
            for (int i = 0; i < 8; i++) {
                topIndex++;
                traversalStack[topIndex + stackIndex] = currNode->deviceChildren[i];
            }
        }
    }
}

int main(int argc, char *argv[]) {
    /* 1 argument
     * FILENAME: The file name to read data from
     */

    if (argc != 2) {
        std::cout << "Incorrect number of parameters given:"
                  << " 1 required, " << argc - 1 << " given" << std::endl
                  << "FILENAME: The file name to read data from" << std::endl;
        return 0;
    }

    // open the file
    std::string filename(argv[1]);
    std::ifstream inFile("../data/" + filename);

    // the number of particles, magnitude of the position,magnitude of the mass
    int N, P, M;
    // inFile >> N >> P >> M;
    // inFile.ignore();
    data >> N >> P >> M;
    // data.ignore();
    // helper variables for reading input
    std::string line;
    double mass;
    double x, y, z;

    bodies.reserve(N);
    for (int i = 0; i < N; i++) {
        // read each body
        // std::getline(inFile, line);

        // create a string stream
        // std::stringstream ss(line);

        data >> mass >> x >> y >> z;

        vec3 position(x, y, z);

        bodies.push_back(new Body(mass, position));
        bodies[i]->velocity = bodies[i]->position / std::pow(10, P - 3);
        bodies[i]->velocity.rot90z();
    }

    std::pair<Node *, CudaNode *> octreePair = CopyTreeToDevice();

    CudaNode *cudaNode_h = new CudaNode();
    CudaNode *cudaNode_d;
    cudaNode_h->mass = 123.0;

    checkCudaErrors(hipMalloc(&cudaNode_d, sizeof(CudaNode)));

    checkCudaErrors(hipMemcpy(cudaNode_d, cudaNode_h, sizeof(CudaNode), hipMemcpyHostToDevice));

    Node *octree = new Node(bodies);
    octree->ConstructTree();
    octree->DFS();
    double *output = (double *)malloc(numNodes * sizeof(double));
    double *output_d;
    checkCudaErrors(hipMalloc((void **)&output_d, sizeof(double) * numNodes));

    dim3 dimBlock(NUM_THREADX, 1, 1);
    dim3 dimGrid((bodies.size() + NUM_THREADX) / NUM_THREADX, 1, 1);

    BarnesHutKernel<<<1, 1, maxStackSize * 1 * sizeof(CudaNode *)>>>(octreePair.second, maxStackSize, output_d);
    // BarnesHutKernel<<<1, 1>>>(octreePair.second, maxStackSize, output_d);
    checkCudaErrors(hipDeviceSynchronize());

    checkCudaErrors(hipMemcpy(output, output_d, sizeof(double) * numNodes, hipMemcpyDeviceToHost));

    for (int i = 0; i < numNodes; i++) {
        DebugPrint(output[i]);
    }

    FreeDeviceTree(octreePair.first, octreePair.second);

    checkCudaErrors(hipFree(output_d));
    delete[] output;

    checkCudaErrors(hipDeviceReset());
}