#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <chrono>
#include <fstream>
#include <iostream>
#include <sstream>
#include <vector>

#include "OpenGL/gl_utilities.h"
#include "utils/hip/hip_runtime_api.h"
#include "utils/node.h"
// #include "utils/utilities.h"s

std::vector<Body *> bodies;
double theta = 0.8;

struct cudaNode {
    long double centreOfMass[3];
    long double mass;
    int children[8];
};

// __global__ void BarnesHutKernel(Node *) {
// }

main(int argc, char *argv[]) {
    /* 1 argument
     * FILENAME: The file name to read data from
     */

    if (argc != 2) {
        std::cout << "Incorrect number of parameters given:"
                  << " 1 required, " << argc - 1 << " given" << std::endl
                  << "FILENAME: The file name to read data from" << std::endl;
        return 0;
    }

    // open the file
    std::string filename(argv[1]);
    std::ifstream inFile("../data/" + filename);

    // the number of particles
    int N;

    // magnitude of the position
    int P;

    // magnitude of the mass
    int M;

    inFile >> N >> P >> M;
    inFile.ignore();

    // helper variables for reading input
    std::string line;
    long double mass;
    long double x;
    long double y;
    long double z;

    bodies.reserve(N);
    for (int i = 0; i < N; i++) {
        // read each body
        std::getline(inFile, line);

        // create a string stream
        std::stringstream ss(line);

        ss >> mass >> x >> y >> z;

        vec3 position(x, y, z);

        bodies.push_back(new Body(mass, position));
        bodies[i]->velocity = bodies[i]->position / std::pow(10, P - 3);
        bodies[i]->velocity.rot90z();
    }
}