#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <chrono>
#include <fstream>
#include <iostream>
#include <sstream>
#include <vector>

#include "OpenGL/gl_utilities.h"
#include "utils/hip/hip_runtime_api.h"
#include "utils/node.h"
#include "utils/utilities.h"

std::vector<Body *> bodies;
ParallelBody *cudaBodies;

double theta = 0.8;
size_t maxStackSize = 0;
int numNodes = 0;

std::stringstream data("6 6 15 9.116473579367844e+15 1.701604548859206e+05 1.305680358210422e+05 -2.130698223755301e+04 8.401877171547095e+15 -2.032763069026615e+04 -1.811363407102262e+05 1.140483239660925e+05 3.943829268190930e+15 -1.791981915863382e+05 5.544912736748732e+03 1.241392662871304e+05 7.830992237586059e+15 5.601020441137095e+03 -8.777286769754969e+04 7.708867393003816e+04 7.984400334760733e+15 3.344990699950628e+04 3.541127793886492e+05 6.195353842703000e+04 1.000000000000000e+19 0.000000000000000e+00 0.000000000000000e+00 0.000000000000000e+00");
int NUM_THREADX = 1;
// Building a separate tree for access on cuda
// preprocessing

template <typename T>
void DebugPrint(T info) {
    std::cout << info << std::endl;
}

__host__ std::pair<Node *, ParallelNode *> CopyTreeToDevice() {
    Node *octree = new Node(bodies);
    octree->ConstructTree();
    // ParallelNode *cudaOctree_h;
    ParallelNode *cudaOctree_d;
    checkCudaErrors(hipMalloc(&cudaOctree_d, sizeof(ParallelNode)));

    std::pair<Node *, ParallelNode *> octree_pair = std::make_pair(octree, cudaOctree_d);

    std::stack<std::pair<Node *, ParallelNode *>> traversalStack;
    traversalStack.push(octree_pair);
    // traverse the original tree
    numNodes++;
    while (!traversalStack.empty()) {
        maxStackSize = std::max(traversalStack.size(), maxStackSize);
        std::pair<Node *, ParallelNode *> currNodePair = traversalStack.top();
        traversalStack.pop();

        if (currNodePair.first->GetNumBodies() < 1) continue;

        ParallelNode *cudaNode_h = new ParallelNode();
        ParallelNode *cudaNode_d = currNodePair.second;

        // copy the bare minimum info needed for barneshut to a Cuda Node

        cudaNode_h->centreOfMass[0] = currNodePair.first->GetCOM(true).x;
        cudaNode_h->centreOfMass[1] = currNodePair.first->GetCOM(true).y;
        cudaNode_h->centreOfMass[2] = currNodePair.first->GetCOM(true).z;
        cudaNode_h->numBodies = currNodePair.first->GetNumBodies();
        cudaNode_h->mass = currNodePair.first->GetMass();
        cudaNode_h->diagDist = std::sqrt(DistSquared(currNodePair.first->GetMaxBound(), currNodePair.first->GetMinBound()));

        if (currNodePair.first->GetNumBodies() != 1) {
            for (int i = 0; i < 8; i++) {
                // allocate each child on the device
                ParallelNode *child_d;
                checkCudaErrors(hipMalloc(&child_d, sizeof(ParallelNode)));

                // keep track of the address of children from the device on host
                cudaNode_h->deviceChildren[i] = child_d;
                currNodePair.first->devicechildren[i] = child_d;
                traversalStack.push(std::make_pair(currNodePair.first->GetChildren()[i], child_d));
                numNodes++;
            }
        };

        // copy the data over
        if (currNodePair.second == cudaOctree_d)
            checkCudaErrors(hipMemcpy(cudaOctree_d, cudaNode_h, sizeof(ParallelNode), hipMemcpyHostToDevice));
        else
            checkCudaErrors(hipMemcpy(cudaNode_d, cudaNode_h, sizeof(ParallelNode), hipMemcpyHostToDevice));

        delete cudaNode_h;
    }

    return octree_pair;
}

void FreeTree(Node *octree, ParallelNode *cudaOctree_d) {
    std::stack<Node *> traversalStack;

    traversalStack.push(octree);

    while (!traversalStack.empty()) {
        Node *currNode = traversalStack.top();
        traversalStack.pop();
        if (currNode->GetNumBodies() <= 1) continue;

        for (int i = 0; i < 8; i++) {
            // clean up gpu memory
            ParallelNode *deviceChild = currNode->devicechildren[i];
            checkCudaErrors(hipFree(deviceChild));

            traversalStack.push(currNode->GetChildren()[i]);
        }
    }

    checkCudaErrors(hipFree(cudaOctree_d));
    delete octree;
}

void UpdateBodies(ParallelBody *cudaBodies) {
    for (int i = 0; i < bodies.size(); i++) {
        bodies[i]->position.x = cudaBodies[i].position[0];
        bodies[i]->position.y = cudaBodies[i].position[1];
        bodies[i]->position.z = cudaBodies[i].position[2];

        bodies[i]->velocity.x = cudaBodies[i].velocity[0];
        bodies[i]->velocity.y = cudaBodies[i].velocity[1];
        bodies[i]->velocity.z = cudaBodies[i].velocity[2];
    }
}

__device__ double DistSquared_d(double *p1, double *p2) {
    double dist = 0.0;
    for (int i = 0; i < 3; i++) {
        dist += (p1[i] - p2[i]) * (p1[i] - p2[i]);
    }
    return dist;
}

__device__ bool isEqual(double one, double two) {
    double eps = 0.1;
    return (fabs(one - two) < eps);
}

__device__ bool SameBody(ParallelBody *body, ParallelNode *node) {
    if (node->numBodies != 1) return false;
    if (!isEqual(body->mass, node->mass)) return false;
    for (int i = 0; i < 3; i++) {
        if (!isEqual(body->position[i], node->centreOfMass[i])) return false;
    }

    return true;
}

__device__ void CalcuateForce(ParallelBody *body, ParallelNode *node, double *totalForce) {
    // TODO: if no work, check G
    double dist = DistSquared_d(body->position, node->centreOfMass);

    double magnitude = G * body->mass * node->mass * (1 / dist);
    double force[3];

    dist = sqrt(dist);
    for (int i = 0; i < 3; i++) {
        force[i] = magnitude * (node->centreOfMass[i] - body->position[i]) / dist;
    }

    for (int i = 0; i < 3; i++)
        totalForce[i] += force[i];
}

__global__ void BarnesHutKernel(ParallelNode *octree_d, size_t maxStackSize_d, ParallelBody *bodies_d, int N, double theta) {
    int threadID = blockIdx.x * blockDim.x + threadIdx.x;

    if (threadID >= N) return;

    ParallelBody *body = (ParallelBody *)bodies_d + threadID;
    double totalForce[3] = {0, 0, 0};

    extern __shared__ ParallelNode *traversalStack[];
    int stackIndex = threadIdx.x * maxStackSize_d;
    int topIndex = -1;  // index of the top element in the stack

    // pushing to stack
    topIndex += 1;
    traversalStack[topIndex + stackIndex] = octree_d;

    int counter = 0;
    // output_d[1] = 420.0;
    while (topIndex != -1) {
        // peeking and popping
        ParallelNode *currNode = traversalStack[topIndex + stackIndex];
        topIndex--;

        if (SameBody(body, currNode)) continue;
        double d = sqrt(DistSquared_d(body->position, currNode->centreOfMass));
        double s = currNode->diagDist;

        if ((s / d) > theta && currNode->numBodies != 1) {
            for (int i = 0; i < 8; i++) {
                ParallelNode *child = currNode->deviceChildren[i];

                if (child->numBodies != 0) {
                    topIndex++;
                    traversalStack[topIndex + stackIndex] = child;
                }
            }
        } else {
            CalcuateForce(body, currNode, totalForce);
        }
    }

    for (int i = 0; i < 3; i++) {
        body->velocity[i] += totalForce[i] / body->mass;
        body->position[i] += body->velocity[i];
    }
}

int main(int argc, char *argv[]) {
    /* 1 argument
     * FILENAME: The file name to read data from
     */

    if (argc != 2) {
        std::cout << "Incorrect number of parameters given:"
                  << " 1 required, " << argc - 1 << " given" << std::endl
                  << "FILENAME: The file name to read data from" << std::endl;
        return 0;
    }

    // open the file
    std::string filename(argv[1]);
    std::ifstream inFile("../data/" + filename);

    // the number of particles, magnitude of the position,magnitude of the mass
    int N, P, M;
    inFile >> N >> P >> M;
    inFile.ignore();
    // data >> N >> P >> M;

    // data.ignore();
    // helper variables for reading input
    std::string line;
    double mass;
    double x, y, z;

    bodies.reserve(N);
    cudaBodies = new ParallelBody[N];
    for (int i = 0; i < N; i++) {
        // read each body
        std::getline(inFile, line);

        // create a string stream
        std::stringstream ss(line);

        ss >> mass >> x >> y >> z;

        vec3 position(x, y, z);

        bodies.push_back(new Body(mass, position));
        bodies[i]->velocity = bodies[i]->position / std::pow(10, P - 3);
        bodies[i]->velocity.rot90z();

        cudaBodies[i].position[0] = position.x;
        cudaBodies[i].position[1] = position.y;
        cudaBodies[i].position[2] = position.z;
        cudaBodies[i].velocity[0] = bodies[i]->velocity.x;
        cudaBodies[i].velocity[1] = bodies[i]->velocity.y;
        cudaBodies[i].velocity[2] = bodies[i]->velocity.z;
        cudaBodies[i].mass = mass;
    }

    // ParallelNode *cudaNode_h = new ParallelNode();
    // ParallelNode *cudaNode_d;
    // cudaNode_h->mass = 123.0;

    // checkCudaErrors(hipMalloc(&cudaNode_d, sizeof(ParallelNode)));
    // checkCudaErrors(hipMemcpy(cudaNode_d, cudaNode_h, sizeof(ParallelNode), hipMemcpyHostToDevice));
    std::pair<Node *, ParallelNode *> octreePair;
    ParallelBody *cudaBodies_d;
    checkCudaErrors(hipMalloc(&cudaBodies_d, sizeof(ParallelBody) * N));

    int numIterations = 100;
    for (int i = 0; i < numIterations; i++) {
        octreePair = CopyTreeToDevice();

        checkCudaErrors(hipMemcpy(cudaBodies_d, cudaBodies, sizeof(ParallelBody) * N, hipMemcpyHostToDevice));

        dim3 dimBlock(NUM_THREADX, 1, 1);
        dim3 dimGrid((bodies.size() + NUM_THREADX) / NUM_THREADX, 1, 1);

        BarnesHutKernel<<<dimGrid, dimBlock, maxStackSize * NUM_THREADX * sizeof(ParallelNode *)>>>(octreePair.second, maxStackSize, cudaBodies_d, N, theta);
        checkCudaErrors(hipDeviceSynchronize());

        checkCudaErrors(hipMemcpy(cudaBodies, cudaBodies_d, sizeof(ParallelBody) * N, hipMemcpyDeviceToHost));
        FreeTree(octreePair.first, octreePair.second);
        UpdateBodies(cudaBodies);
				std::cout<<i<<std::endl;
    }

    std::ofstream outFile("../out/cuda-" + filename);
    outFile.precision(15);

    outFile << N << std::endl;
    outFile << std::scientific;
    for (Body *body : bodies) {
        outFile << body->mass
                << " " << body->position.x
                << " " << body->position.y
                << " " << body->position.z
                << std::endl;
        delete body;
    }

    outFile.close();

    checkCudaErrors(hipFree(cudaBodies_d));
    delete[] cudaBodies;
    checkCudaErrors(hipDeviceReset());
}